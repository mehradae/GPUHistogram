
#include <hip/hip_runtime.h>
/******************************************************************************
 *cr
 *cr            (C) Copyright 2010 The Board of Trustees of the
 *cr                        University of Illinois
 *cr                         All Rights Reserved
 *cr
 ******************************************************************************/

// Define your kernels in this file you may use more than one kernel if you
// need to

const unsigned int BLOCK_SIZE = 512;


__global__ void Histo_Kernel(unsigned int* input, unsigned int* bins, unsigned int num_elements,
                             unsigned int num_bins) {


    int index = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = blockDim.x * gridDim.x;

    //using shared memory
    extern __shared__ unsigned int histo_private[];
    for(int i = threadIdx.x; i < num_bins; i += blockDim.x) {
        histo_private[i] = 0;
    }

    __syncthreads();


    while (index < num_elements) {
//        atomicAdd(&(histo_private[(input[index])]), 1);
        atomicAdd(&(histo_private[(input[index])]), 1);
        index += stride;
    }
    __syncthreads();

    //create final histogram using atomic add
    for(int j = threadIdx.x; j < num_bins; j += blockDim.x) {
        atomicAdd(&(bins[j]), histo_private[j]);
    }
    __syncthreads();

}


/******************************************************************************
Setup and invoke your kernel(s) in this function. You may also allocate more
GPU memory if you need to
*******************************************************************************/
void histogram(unsigned int* input, unsigned int* bins, unsigned int num_elements,
        unsigned int num_bins) {

    Histo_Kernel<<<ceil(num_elements/BLOCK_SIZE), BLOCK_SIZE,
            sizeof(unsigned int)*num_bins>>>(input, bins, num_elements, num_bins);

}


